#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "deviceQuery.h"
#include <tuple>

constexpr int WarpsStart = 1;
constexpr int WarpsEnd = 32768;   // maxium number of warps on GPU

constexpr int ThreadsPerWarp = 32;
constexpr int IterCount = 1024;   // number of iterations of pointer chasing per thread.


constexpr int ThreadsMax = WarpsEnd * ThreadsPerWarp;
// maximum size of pointer-chasing array. 32 * 1024 * 32768 = 1GB
// Make sure ArraySizeMax > 2 * L2_SIZE 
constexpr uint64_t ArraySizeMax = IterCount * ThreadsPerWarp * WarpsEnd; 

constexpr int MaxWarpPerBlock = 32;


struct Param_Struct {
  Param_Struct() {
    startClk = (uint32_t *)malloc(WarpsEnd * sizeof(uint32_t));
    stopClk = (uint32_t *)malloc(WarpsEnd * sizeof(uint32_t));
    dsink = (uint64_t *)malloc(WarpsEnd * sizeof(uint64_t));

    gpuErrchk(hipMalloc(&startClk_g, WarpsEnd * sizeof(uint32_t)));
    gpuErrchk(hipMalloc(&stopClk_g, WarpsEnd * sizeof(uint32_t)));
    gpuErrchk(hipMalloc(&dsink_g, WarpsEnd * sizeof(uint64_t)));
  }

  ~Param_Struct() {
    free(startClk);
    free(stopClk);
    free(dsink);

    gpuErrchk(hipFree(startClk_g));
    gpuErrchk(hipFree(stopClk_g));
    gpuErrchk(hipFree(dsink_g));
  }

  void memcpyd2h() {
    gpuErrchk(hipMemcpy(startClk, startClk_g, WarpsEnd * sizeof(uint32_t),
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(stopClk, stopClk_g, WarpsEnd * sizeof(uint32_t),
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(dsink, dsink_g, WarpsEnd * sizeof(uint64_t),
                          hipMemcpyDeviceToHost));
  }
  uint32_t getDuration(){
    uint32_t duration = 0;
    for(int i = 0; i < WarpsEnd; i++)
    {
      uint32_t temp = stopClk[i] - startClk[i];
      if(temp > duration)
        duration = temp;
    }
    return duration;
  }

  uint32_t *startClk;
  uint32_t *stopClk;
  uint64_t *dsink;

  uint32_t *startClk_g;
  uint32_t *stopClk_g;
  uint64_t *dsink_g;
};

__global__ void mem_lat(uint64_t *pointer_chasing_array, uint32_t *startClk_g, uint32_t *stopClk_g, uint64_t *dsink_g) {
  // thread index
  uint32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
  uint32_t uid = blockIdx.x * blockDim.x * blockDim.y + tid;
  uint32_t wid = uid / ThreadsPerWarp;

  uint64_t ptr = (uint64_t)pointer_chasing_array + uid * sizeof(uint64_t); 
  uint64_t ptr1 = 0, ptr0 = 0;

  uint32_t start = 0;
  // start timing
  asm volatile("mov.u32 %0, %%clock;" : "=r"(start)::"memory");

  // initialize the pointers with the start address
  // Here, we use cache volatile modifier to ignore the L2 cache
  // TODO on Ampere it seems .cv modifier is not effective
  asm volatile("{\t\n"
                "ld.global.cv.u64 %0, [%1];\n\t"
                "}"
                : "=l"(ptr1)
                : "l"(ptr)
                : "memory");

  // pointer-chasing IterCount times
  // Here, we use cache volatile modifier to ignore the L2 cache
  for (uint32_t i = 0; i < IterCount - 1; i++) {
    asm volatile("{\t\n"
                  "ld.global.cv.u64 %0, [%1];\n\t"
                  "}"
                  : "=l"(ptr0)
                  : "l"(ptr1)
                  : "memory");
    ptr1 = ptr0; // swap the register for the next load
    // if(uid == 0) printf("Step %d\n", i);
    // printf("uid = %d, ptr0 = %lx, ptr1 = %lx\n", uid, ptr0, ptr1);
  }
  // TODO bar.sync is used to avoid compiler rearranging the move clock before ld.global. If there other ways to do this?
  // synchronize all threads
  asm volatile("bar.sync 0;");

  uint32_t stop = 0;
  // stop timing
  asm volatile("mov.u32 %0, %%clock;" : "=r"(stop)::"memory");

  // write time and data back to memory
  if(uid % ThreadsPerWarp == 0)
  {
    // printf("uid = %d, start = %u, stop = %u\n", uid, start, stop);
    startClk_g[wid] = start;
    stopClk_g[wid] = stop;
    dsink_g[wid] = ptr1;
  }
}

// TODO this function has bug. It is replaced by CPU initialization
// Initialize pointer chasing array of ArraySizeMax in global memory with ThreadsMax threads
// __global__ void initPointerArray(uint64_t * array)
// {
//   // thread index
//   uint32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
//   uint32_t uid = blockIdx.x * blockDim.x * blockDim.y + tid;
//   // initialize pointer-chasing array by CPU to avoid messing up nsight-compute counters

//   for(int i = 0; i < (IterCount - 1); i += 1)
//     array[i * ThreadsMax + uid] = (uint64_t)array + (i + ThreadsMax) * sizeof(uint64_t);
//   array[(IterCount - 1) * ThreadsMax + uid] = (uint64_t)array + uid * sizeof(uint64_t);
// }

template <int NumWarps>
void measureMemLat (Param_Struct & param) {
  unsigned array_size =  ArraySizeMax; 
  uint64_t *pointer_chasing_array_g;
  gpuErrchk(hipMalloc(&pointer_chasing_array_g, array_size * sizeof(uint64_t)));

  // initPointerArray<<<WarpsEnd/MaxWarpPerBlock, ThreadsPerWarp * MaxWarpPerBlock>>>(pointer_chasing_array_g);
  // gpuErrchk(hipPeekAtLastError());

  // initialize pointer-chasing array by CPU to avoid messing up nsight-compute counters
  uint64_t * pointer_chasing_array = (uint64_t *)malloc(array_size * sizeof(uint64_t));
  for (uint32_t i = 0; i < (array_size - ThreadsMax); i += 1)
    pointer_chasing_array[i] = (uint64_t)pointer_chasing_array_g + (i + ThreadsMax) * sizeof(uint64_t);
  // initialize the tail to reference to the head of the array
  for (uint32_t i = (array_size - ThreadsMax); i < array_size; i += 1)
    pointer_chasing_array[i] = (uint64_t)pointer_chasing_array_g + (i - (array_size - ThreadsMax)) * sizeof(uint64_t);
  gpuErrchk(hipMemcpy(pointer_chasing_array_g, pointer_chasing_array, array_size * sizeof(uint64_t),
                       hipMemcpyHostToDevice));
    
  // kernel launch
  dim3 grid(NumWarps > MaxWarpPerBlock ? NumWarps / MaxWarpPerBlock : 1);
  dim3 block(ThreadsPerWarp, NumWarps > MaxWarpPerBlock ? MaxWarpPerBlock : NumWarps);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  mem_lat<<<grid, block>>>(pointer_chasing_array_g, param.startClk_g, param.stopClk_g, param.dsink_g);

  gpuErrchk(hipPeekAtLastError());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  param.memcpyd2h();

  float lat = (float)(param.getDuration()) / (float)(IterCount);
  float bw = (float)(IterCount * NumWarps * ThreadsPerWarp * sizeof(uint64_t)) / (float)(milliseconds * 1e6);
  int outstanding_bytes = NumWarps * ThreadsPerWarp * sizeof(uint64_t);
  printf("%d,%f,%f\n", outstanding_bytes, lat, bw);
  // printf("INFO: Measuring memory latency with %d warps\n", NumWarps); 
  // printf("Mem latency = %12.4f cycles \n", lat);
  // printf("Kernel time = %f ms\n", milliseconds);
  // printf("Mem Bandwidth = %f GB/s \n", bw);
  // printf("Clk number per Warp= %u \n", param.getDuration());
  gpuErrchk(hipFree(pointer_chasing_array_g));
  free(pointer_chasing_array);
}

template <int NumWarps>
struct MemLatFunction {
  static void call(Param_Struct &param) {
    measureMemLat<NumWarps>(param);
  }
};


template <int WarpsStart, int WarpsEnd>
struct GenerateMemLat {
  static auto generate() {
    if constexpr (WarpsStart < WarpsEnd) {
          return std::tuple_cat(
              std::tuple<void (*)(Param_Struct &)>{
                &MemLatFunction<WarpsStart>::call},
              GenerateMemLat<WarpsStart * 2, WarpsEnd>::generate());
    } else {
      return std::tuple<>();
    }
  }
};

template <typename... Functions, std::size_t... Is>
void callAllFunctionsImpl(const std::tuple<Functions...> &functionList,
                          Param_Struct &param,
                          std::index_sequence<Is...>) {
  ((std::get<Is>(functionList))(param), ...);
}

template <typename... Functions>
void callAllFunctions(const std::tuple<Functions...> &functionList,
                      Param_Struct &param) {
  callAllFunctionsImpl(functionList, param,
                       std::index_sequence_for<Functions...>{});
}

int main() {

  intilizeDeviceProp(0);
  Param_Struct param;
  auto functionList = GenerateMemLat<WarpsStart, WarpsEnd>::generate();
  printf("OutstandingRequests(B),Latency(ns),Bandwidth(GB/s)\n");
  callAllFunctions(functionList, param);

  return 0;
}
